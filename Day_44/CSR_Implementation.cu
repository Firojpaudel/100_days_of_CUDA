#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip> // For output formatting

using namespace std;

// Define CSR Matrix Structure
struct CSRMatrix {
    int numRows;          // Number of rows
    int numCols;          // Number of columns
    int numNonzeros;      // Number of nonzero elements
    unsigned int* rowPtrs; // Pointers to the start of each row
    unsigned int* colIdx;  // Column indices of non-zero elements
    float* value;         // Values of non-zero elements
};

// CUDA error check macro
#define CUDA_CHECK(call)                                                        \
    do {                                                                        \
        hipError_t error = call;                                               \
        if (error != hipSuccess) {                                             \
            printf("CUDA error %04d: %s file: %s line: %d\n", error,            \
                   hipGetErrorString(error), __FILE__, __LINE__);              \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

// Utility function to print an array on the host
void printArray(float* arr, int n, const char* label) {
    cout << "\n" << label << ":\n";
    cout << fixed << setprecision(2); // 2 decimal places for floats
    for (int i = 0; i < n; i++) {
        cout << "  [" << setw(2) << i << "] = " << setw(8) << arr[i];  //setw(2) -> setwidth(2) using this for clean output print formatting :) 
        if (i < n - 1) cout << "\n"; // Newline except for last element 😁
    }
    cout << endl;
}

// Function to print CSR matrix in dense format
void printCSRMatrixDense(int numRows, int numCols, int numNonzeros, unsigned int* rowPtrs, unsigned int* colIdx, float* value) {
    cout << "\nSparse Matrix A in dense format:\n";
    cout << fixed << setprecision(2); // Consistent float formatting
    for (int i = 0; i < numRows; i++) {
        int idx = rowPtrs[i]; // Start index for current row
        for (int j = 0; j < numCols; j++) {
            if (idx < rowPtrs[i + 1] && colIdx[idx] == j) {
                cout << setw(8) << value[idx];  
                idx++; // Move to next non-zero element in the row
            } else {
                cout << setw(8) << 0.00; // Print zero for empty positions
            }
        }
        cout << endl; // New line after each row
    }
}

// Function to print the calculation process for y = A * x
void printCalculationProcess(int numRows, int numNonzeros, unsigned int* rowPtrs, unsigned int* colIdx, float* value, float* x, float* y) {
    cout << "\nCalculation process for y = A * x:\n";
    cout << fixed << setprecision(2); // Consistent float formatting
    for (int i = 0; i < numRows; i++) {
        cout << "y[" << i << "] = ";
        bool first = true; // Flag for formatting terms
        float sum = 0.0f;  // Host-computed sum for this row
        for (int idx = rowPtrs[i]; idx < rowPtrs[i + 1]; idx++) {
            int j = colIdx[idx];
            float val = value[idx];
            float term = val * x[j];
            sum += term;
            if (!first) cout << " + ";
            cout << val << " * " << x[j] << " (A[" << i << "][" << j << "] * x[" << j << "])";
            first = false;
        }
        cout << " = " << sum << " (Host) vs " << y[i] << " (GPU)" << endl;
    }
}

// CUDA Kernel for SpMV with CSR (provided by you)
__global__ void spmv_csr_kernel(CSRMatrix csrMatrix, float* x, float* y) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < csrMatrix.numRows) {
        float sum = 0.0f;
        for (unsigned int i = csrMatrix.rowPtrs[row]; i < csrMatrix.rowPtrs[row + 1]; ++i) {
            unsigned int col = csrMatrix.colIdx[i];
            float value = csrMatrix.value[i];
            sum += x[col] * value;
        }
        y[row] += sum;
    }
}

int main() {
    // Define matrix dimensions and number of non-zeros
    int numRows = 4;
    int numCols = 4;
    int numNonzeros = 8;

    // Dynamically allocate host arrays for CSR format
    unsigned int* h_rowPtrs = new unsigned int[numRows + 1]{0, 2, 5, 7, 8}; // One extra for the end
    unsigned int* h_colIdx = new unsigned int[numNonzeros]{0, 1, 0, 2, 3, 1, 2, 3};
    float* h_value = new float[numNonzeros]{1.0f, 7.0f, 5.0f, 3.0f, 9.0f, 2.0f, 8.0f, 6.0f};
    float* h_x = new float[numCols]{1.0f, 2.0f, 3.0f, 4.0f};  // Input vector
    float* h_y = new float[numRows]{0.0f, 0.0f, 0.0f, 0.0f};  // Output vector

    // Step 1: Display the sparse matrix A
    printCSRMatrixDense(numRows, numCols, numNonzeros, h_rowPtrs, h_colIdx, h_value);

    // Device arrays
    unsigned int *d_rowPtrs, *d_colIdx;
    float *d_value, *d_x, *d_y;

    // Allocate memory on the device
    CUDA_CHECK(hipMalloc(&d_rowPtrs, (numRows + 1) * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_colIdx, numNonzeros * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_value, numNonzeros * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_x, numCols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, numRows * sizeof(float)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_rowPtrs, h_rowPtrs, (numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_colIdx, h_colIdx, numNonzeros * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_value, h_value, numNonzeros * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, numCols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, numRows * sizeof(float), hipMemcpyHostToDevice));

    // Create CSRMatrix object on the host
    CSRMatrix csrMatrix;
    csrMatrix.numRows = numRows;
    csrMatrix.numCols = numCols;
    csrMatrix.numNonzeros = numNonzeros;
    csrMatrix.rowPtrs = d_rowPtrs;
    csrMatrix.colIdx = d_colIdx;
    csrMatrix.value = d_value;

    // Set block and grid size
    int blockSize = 256; 
    int numBlocks = (numRows + blockSize - 1) / blockSize; // Based on rows, not non-zeros

    // Launch kernel
    spmv_csr_kernel<<<numBlocks, blockSize>>>(csrMatrix, d_x, d_y);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result from device to host
    CUDA_CHECK(hipMemcpy(h_y, d_y, numRows * sizeof(float), hipMemcpyDeviceToHost));

    // Step 2: Display input vector, calculation process, and output
    printArray(h_x, numCols, "Input Vector x");
    printCalculationProcess(numRows, numNonzeros, h_rowPtrs, h_colIdx, h_value, h_x, h_y);
    printArray(h_y, numRows, "Output Vector y");

    // Free memory on device
    CUDA_CHECK(hipFree(d_rowPtrs));
    CUDA_CHECK(hipFree(d_colIdx));
    CUDA_CHECK(hipFree(d_value));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    // Free memory on host
    delete[] h_rowPtrs;
    delete[] h_colIdx;
    delete[] h_value;
    delete[] h_x;
    delete[] h_y;

    return 0;
}