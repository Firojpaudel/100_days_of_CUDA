#include <hip/hip_runtime.h>
#include <float.h>

__global__ void minReduceKernel(
    const float* __restrict__ input,
    float* __restrict__ output,
    size_t before_size,
    size_t dim_size,
    size_t after_size,
    int dim,
    size_t ndim,
    const size_t* __restrict__ shape
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t output_size = before_size * after_size;
    
    if (idx >= output_size) return;
    
    size_t before_idx = idx / after_size;
    size_t after_idx = idx % after_size;
    
    float min_val = FLT_MAX;
    
    // Special case: Reducing the last dimension with contiguous memory
    if (dim == ndim - 1 && after_size == 1) {
        size_t start_idx = before_idx * dim_size;
        // Use float4 for coalesced 128-bit reads
        size_t d = 0;
        for (; d + 3 < dim_size; d += 4) {
            float4 val = *reinterpret_cast<const float4*>(&input[start_idx + d]);
            min_val = min(min_val, min(min(val.x, val.y), min(val.z, val.w)));
        }
        // Handle remaining elements
        for (; d < dim_size; d++) {
            min_val = min(min_val, input[start_idx + d]);
        }
    } else {
        // General case: Strided access
        for (size_t d = 0; d < dim_size; d++) {
            size_t input_idx = (before_idx * dim_size + d) * after_size + after_idx;
            min_val = min(min_val, input[input_idx]);
        }
    }
    
    output[idx] = min_val;
}

extern "C" void solution(
    const float* input,
    int dim,
    float* output,
    size_t* shape,
    size_t ndim
) {
    size_t before_size = 1;
    for (int i = 0; i < dim; i++) {
        before_size *= shape[i];
    }
    
    size_t dim_size = shape[dim];
    
    size_t after_size = 1;
    for (int i = dim + 1; i < ndim; i++) {
        after_size *= shape[i];
    }
    
    size_t output_size = before_size * after_size;
    
    int blockSize = 256;
    int numBlocks = (output_size + blockSize - 1) / blockSize;
    
    minReduceKernel<<<numBlocks, blockSize>>>(
        input, output, before_size, dim_size, after_size, dim, ndim, shape
    );
}