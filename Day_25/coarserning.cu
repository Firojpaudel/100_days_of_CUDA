#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
using namespace std;

#define NUM_BINS 26
#define COARSENING_FACTOR 4  // Each thread processes 4 elements

// CUDA kernel using Contiguous Partitioning for Coarsening
__global__ void histo_coarsening_contiguous(char* data, unsigned int length, unsigned int* histo) {
    __shared__ unsigned int histo_s[NUM_BINS];

    // Initialize shared memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread processes COARSENING_FACTOR consecutive elements
    for (unsigned int i = tid * COARSENING_FACTOR; i < length; i += blockDim.x * gridDim.x * COARSENING_FACTOR) {
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            if (i + j < length) {
                int alphabet_position = data[i + j] - 'a';
                if (alphabet_position >= 0 && alphabet_position < 26) {
                    atomicAdd(&(histo_s[alphabet_position]), 1);
                }
            }
        }
    }
    __syncthreads();

    // Merge to global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        atomicAdd(&(histo[bin]), histo_s[bin]);
    }
}

// CUDA kernel using Interleaved Partitioning for Coarsening
__global__ void histo_coarsening_interleaved(char* data, unsigned int length, unsigned int* histo) {
    __shared__ unsigned int histo_s[NUM_BINS];

    // Initialize shared memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        histo_s[bin] = 0u;
    }
    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread processes COARSENING_FACTOR elements spaced apart
    for (unsigned int i = tid; i < length; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < COARSENING_FACTOR; j++) {
            unsigned int index = i + j * (blockDim.x * gridDim.x);
            if (index < length) {
                int alphabet_position = data[index] - 'a';
                if (alphabet_position >= 0 && alphabet_position < 26) {
                    atomicAdd(&(histo_s[alphabet_position]), 1);
                }
            }
        }
    }
    __syncthreads();

    // Merge to global memory
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        atomicAdd(&(histo[bin]), histo_s[bin]);
    }
}

// Main function
int main() {
    cout << "Enter a string (lowercase letters only): ";
    string input;
    getline(cin, input);
    unsigned int length = input.length();

    // Allocate and copy data to device
    char* d_data;
    hipMalloc((void**)&d_data, length * sizeof(char));
    hipMemcpy(d_data, input.c_str(), length * sizeof(char), hipMemcpyHostToDevice);

    // Allocate histogram on device
    unsigned int* d_histo_contiguous;
    unsigned int* d_histo_interleaved;
    hipMalloc((void**)&d_histo_contiguous, NUM_BINS * sizeof(unsigned int));
    hipMalloc((void**)&d_histo_interleaved, NUM_BINS * sizeof(unsigned int));
    hipMemset(d_histo_contiguous, 0, NUM_BINS * sizeof(unsigned int));
    hipMemset(d_histo_interleaved, 0, NUM_BINS * sizeof(unsigned int));

    int blockSize = 256;
    int gridSize = (length + blockSize * COARSENING_FACTOR - 1) / (blockSize * COARSENING_FACTOR);

    // Timing events
    hipEvent_t start_contiguous, stop_contiguous;
    hipEvent_t start_interleaved, stop_interleaved;
    hipEventCreate(&start_contiguous);
    hipEventCreate(&stop_contiguous);
    hipEventCreate(&start_interleaved);
    hipEventCreate(&stop_interleaved);

    // Launch the Contiguous Partitioning kernel
    hipEventRecord(start_contiguous);
    histo_coarsening_contiguous<<<gridSize, blockSize>>>(d_data, length, d_histo_contiguous);
    hipEventRecord(stop_contiguous);
    hipEventSynchronize(stop_contiguous);
    float milliseconds_contiguous = 0;
    hipEventElapsedTime(&milliseconds_contiguous, start_contiguous, stop_contiguous);

    // Launch the Interleaved Partitioning kernel
    hipEventRecord(start_interleaved);
    histo_coarsening_interleaved<<<gridSize, blockSize>>>(d_data, length, d_histo_interleaved);
    hipEventRecord(stop_interleaved);
    hipEventSynchronize(stop_interleaved);
    float milliseconds_interleaved = 0;
    hipEventElapsedTime(&milliseconds_interleaved, start_interleaved, stop_interleaved);

    // Copy back results for both kernels
    unsigned int h_histo_contiguous[NUM_BINS];
    unsigned int h_histo_interleaved[NUM_BINS];
    hipMemcpy(h_histo_contiguous, d_histo_contiguous, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(h_histo_interleaved, d_histo_interleaved, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Display results
    cout << "\nCharacter Frequency Histogram (Contiguous Partitioning):\n";
    for (int i = 0; i < NUM_BINS; ++i) {
        if (h_histo_contiguous[i] > 0) {
            cout << char('a' + i) << ": " << h_histo_contiguous[i] << endl;
        }
    }
    cout << "\nKernel Execution Time (Contiguous Partitioning): " << milliseconds_contiguous << " ms\n";

    cout << "\nCharacter Frequency Histogram (Interleaved Partitioning):\n";
    for (int i = 0; i < NUM_BINS; ++i) {
        if (h_histo_interleaved[i] > 0) {
            cout << char('a' + i) << ": " << h_histo_interleaved[i] << endl;
        }
    }
    cout << "\nKernel Execution Time (Interleaved Partitioning): " << milliseconds_interleaved << " ms\n";

    // Cleanup
    hipFree(d_data);
    hipFree(d_histo_contiguous);
    hipFree(d_histo_interleaved);
    hipEventDestroy(start_contiguous);
    hipEventDestroy(stop_contiguous);
    hipEventDestroy(start_interleaved);
    hipEventDestroy(stop_interleaved);

    return 0;
}