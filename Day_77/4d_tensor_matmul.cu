#include <hip/hip_runtime.h>

__global__ void tensorMatrixMultKernel(
    const float* A,
    const float* B,
    float* C,
    size_t B_dim,
    size_t I_dim,
    size_t J_dim,
    size_t L_dim,
    size_t K_dim
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = B_dim * I_dim * J_dim * K_dim;

    if (idx < total_elements) {
        int k = idx % K_dim;
        int j = (idx / K_dim) % J_dim;
        int i = (idx / (K_dim * J_dim)) % I_dim;
        int b = idx / (K_dim * J_dim * I_dim);

        size_t c_idx = ((b * I_dim + i) * J_dim + j) * K_dim + k;
        size_t a_base = ((b * I_dim + i) * J_dim + j) * L_dim;

        float sum = 0.0f;

        // Vectorized load for A using float2
        for (int l = 0; l < L_dim - 1; l += 2) {
            float2 a_vals = *reinterpret_cast<const float2*>(&A[a_base + l]);
            sum += a_vals.x * B[l * K_dim + k];
            sum += a_vals.y * B[(l + 1) * K_dim + k];
        }
        // Handle remaining element if L_dim is odd
        if (L_dim % 2 == 1) {
            sum += A[a_base + L_dim - 1] * B[(L_dim - 1) * K_dim + k];
        }

        C[c_idx] = sum;
    }
}

extern "C" void solution(
    const float* A,
    const float* B,
    float* C,
    size_t b,
    size_t i,
    size_t j,
    size_t l,
    size_t k
) {
    size_t total_elements = b * i * j * k;
    int threadsPerBlock = 256;
    int blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
    tensorMatrixMultKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, b, i, j, l, k);
}