#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define SECTION_SIZE 1024 // Number of elements per block (adjustable)

// Optimized Kogge-Stone Inclusive Scan Kernel
__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int N) {
    __shared__ float XY[SECTION_SIZE]; // Shared memory for the block

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global index

    // Load data into shared memory
    if (i < N) {
        XY[threadIdx.x] = X[i];
    } else {
        XY[threadIdx.x] = 0.0f; // Padding with identity value
    }
    __syncthreads();

    // Perform Kogge-Stone scan within each block
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        if (threadIdx.x >= stride) {
            XY[threadIdx.x] += XY[threadIdx.x - stride];
        }
        __syncthreads(); // Synchronize after updating shared memory
    }

    // Write results back to global memory
    if (i < N) {
        Y[i] = XY[threadIdx.x];
    }
}

int main() {
    const int N = 16;
    float h_X[N];
    float h_Y[N];

    float *d_X, *d_Y;

    // Taking user input for the array
    cout << "Enter " << N << " elements for the input array:\n";
    for (int i = 0; i < N; ++i) {
        cin >> h_X[i];
    }

    // Allocating device memory
    hipMalloc((void**)&d_X, N * sizeof(float));
    hipMalloc((void**)&d_Y, N * sizeof(float));

    // Copying input data to device
    hipMemcpy(d_X, h_X, N * sizeof(float), hipMemcpyHostToDevice);

    // Launching kernel
    int threadsPerBlock = min(N, SECTION_SIZE);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Start the timer
    auto start = high_resolution_clock::now();

    Kogge_Stone_scan_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_X, d_Y, N);

    // Ensure the kernel has finished before stopping the timer
    hipDeviceSynchronize();

    // Stop the timer
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);

    // Copying results back to host
    hipMemcpy(h_Y, d_Y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Printing results
    cout << "Input Array: ";
    for (int i = 0; i < N; ++i) {
        cout << h_X[i] << " ";
    }
    cout << "\n";

    cout << "Output Array (Inclusive Scan): ";
    for (int i = 0; i < N; ++i) {
        cout << h_Y[i] << " ";
    }
    cout << "\n";

    cout << "Execution time: " << duration.count() / 1000.0f << " ms" << endl;

    // Freeing device memory
    hipFree(d_X);
    hipFree(d_Y);

    return 0;
}