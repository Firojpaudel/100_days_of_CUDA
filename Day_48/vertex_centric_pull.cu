#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <climits>
using namespace std;

#define CUDA_CHECK(call)                                           \
    do                                                             \
    {                                                              \
        hipError_t err = call;                                    \
        if (err != hipSuccess)                                    \
        {                                                          \
            cerr << "CUDA Error: " << hipGetErrorString(err)      \
                 << " at " << __FILE__ << ":" << __LINE__ << endl; \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    } while (0)

struct CSRGraph
{
    unsigned int numVertices;
    unsigned int *srcPtrs;
    unsigned int *dst;
};

__global__ void bfs_bottom_up_kernel(CSRGraph csrGraph, unsigned int *level,
                                     unsigned int *newVertexVisited, unsigned int currLevel)
{
    unsigned int vertex = blockIdx.x * blockDim.x + threadIdx.x;
    if (vertex < csrGraph.numVertices)
    {
        if (level[vertex] == UINT_MAX)
        {
            for (unsigned int edge = csrGraph.srcPtrs[vertex];
                 edge < csrGraph.srcPtrs[vertex + 1]; ++edge)
            {
                unsigned int neighbor = csrGraph.dst[edge];
                if (level[neighbor] == currLevel - 1)
                {
                    level[vertex] = currLevel;
                    atomicExch(newVertexVisited, 1U);
                    break;
                }
            }
        }
    }
}

int main()
{
    unsigned int numVertices = 6;
    vector<unsigned int> srcPtrs_host = {0, 2, 3, 4, 4, 4, 4};
    vector<unsigned int> dst_host = {1, 2, 3, 4};
    // Transpose graph for bottom-up BFS
    vector<unsigned int> srcPtrs_trans_host = {0, 0, 1, 2, 3, 4, 4};
    vector<unsigned int> dst_trans_host = {0, 0, 1, 2};

    cout << "Starting BFS on graph with " << numVertices << " vertices." << endl;

    // Device memory allocation
    unsigned int *d_srcPtrs, *d_dst, *d_level, *d_newVertexVisited;
    unsigned int *d_srcPtrs_trans, *d_dst_trans;
    CUDA_CHECK(hipMalloc(&d_srcPtrs, (numVertices + 1) * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_dst, dst_host.size() * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_srcPtrs_trans, (numVertices + 1) * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_dst_trans, dst_trans_host.size() * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_level, numVertices * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_newVertexVisited, sizeof(unsigned int)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_srcPtrs, srcPtrs_host.data(),
                          (numVertices + 1) * sizeof(unsigned int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dst, dst_host.data(),
                          dst_host.size() * sizeof(unsigned int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_srcPtrs_trans, srcPtrs_trans_host.data(),
                          (numVertices + 1) * sizeof(unsigned int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dst_trans, dst_trans_host.data(),
                          dst_trans_host.size() * sizeof(unsigned int),
                          hipMemcpyHostToDevice));

    // Initialize levels
    CUDA_CHECK(hipMemset(d_level, 0xFF, numVertices * sizeof(unsigned int)));
    unsigned int zero = 0;
    unsigned int start = 0;
    CUDA_CHECK(hipMemcpy(&d_level[start], &zero, sizeof(unsigned int),
                          hipMemcpyHostToDevice));

    // Configure transpose CSRGraph
    CSRGraph transGraph;
    transGraph.numVertices = numVertices;
    transGraph.srcPtrs = d_srcPtrs_trans;
    transGraph.dst = d_dst_trans;

    // BFS loop
    unsigned int currLevel = 1;
    bool continueBFS = true;
    while (continueBFS)
    {
        cout << "Processing level " << currLevel << "..." << endl;
        CUDA_CHECK(hipMemset(d_newVertexVisited, 0, sizeof(unsigned int)));

        int blockSize = 256;
        int gridSize = (numVertices + blockSize - 1) / blockSize;
        bfs_bottom_up_kernel<<<gridSize, blockSize>>>(transGraph, d_level,
                                                      d_newVertexVisited, currLevel);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        unsigned int newVisited;
        CUDA_CHECK(hipMemcpy(&newVisited, d_newVertexVisited,
                              sizeof(unsigned int), hipMemcpyDeviceToHost));
        if (newVisited == 0)
        {
            cout << "No new vertices visited at level " << currLevel
                 << ", BFS complete." << endl;
            continueBFS = false;
        }
        else
        {
            cout << "New vertices visited, moving to level " << currLevel + 1
                 << "." << endl;
            currLevel++;
        }
    }

    // Copy results to host
    vector<unsigned int> level_host(numVertices);
    CUDA_CHECK(hipMemcpy(level_host.data(), d_level,
                          numVertices * sizeof(unsigned int),
                          hipMemcpyDeviceToHost));

    // Print results
    cout << "\nBFS Results:" << endl;
    for (unsigned int i = 0; i < numVertices; ++i)
    {
        cout << "Vertex " << i << ": Level ";
        if (level_host[i] == UINT_MAX)
        {
            cout << "Unreachable" << endl;
        }
        else
        {
            cout << level_host[i] << endl;
        }
    }

    // Free memory
    CUDA_CHECK(hipFree(d_srcPtrs));
    CUDA_CHECK(hipFree(d_dst));
    CUDA_CHECK(hipFree(d_srcPtrs_trans));
    CUDA_CHECK(hipFree(d_dst_trans));
    CUDA_CHECK(hipFree(d_level));
    CUDA_CHECK(hipFree(d_newVertexVisited));

    return 0;
}