#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define BLOCK_DIM 1024
#define COARSE_FACTOR 2  // Defining coarsening factor

// Defining the kernel for performing coarsened sum reduction using shared memory
__global__ void CoarsenedSumReductionKernel(float* input, float* output, int numElements) {
    __shared__ float input_s[BLOCK_DIM];
    unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;
    unsigned int t = threadIdx.x;

    // Checking boundary condition
    if (i >= numElements) return;

    // Performing coarsened sum reduction
    float sum = (i < numElements) ? input[i] : 0.0f;
    for (unsigned int tile = 1; tile < COARSE_FACTOR * 2; ++tile) {
        unsigned int idx = i + tile * BLOCK_DIM;
        if (idx < numElements) {
            sum += input[idx];
        }
    }

    // Storing partial sum in shared memory
    input_s[t] = sum;
    __syncthreads();

    // Performing reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        if (t < stride) {
            input_s[t] += input_s[t + stride];
        }
        __syncthreads();
    }

    // Using atomic add to combine partial sums
    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}

int main() {
    // Asking the user for the number of elements
    int numElements;
    cout << "Enter the number of elements: ";
    cin >> numElements;

    int size = numElements * sizeof(float);

    // Allocating host memory
    float* h_input = new float[numElements];
    float h_output = 0.0f;

    // Initializing input data
    for (int i = 0; i < numElements; i++) {
        h_input[i] = 1.0f; // Assigning all elements to 1.0 for easy summing
    }

    // Allocating device memory
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, sizeof(float));

    // Copying input data to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, &h_output, sizeof(float), hipMemcpyHostToDevice);

    // Calculating grid and block sizes
    int numBlocks = (numElements + (COARSE_FACTOR * 2 * BLOCK_DIM) - 1) / (COARSE_FACTOR * 2 * BLOCK_DIM);

    // Measuring execution time
    auto start = high_resolution_clock::now();
    
    // Launching the kernel
    CoarsenedSumReductionKernel<<<numBlocks, BLOCK_DIM>>>(d_input, d_output, numElements);
    
    // Synchronizing device
    hipDeviceSynchronize();
    
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start);

    // Copying the result back to host
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Printing the final sum and execution time
    cout << "Final sum: " << h_output << endl;
    cout << "Execution time: " << duration.count() << " ms" << endl;

    // Freeing allocated memory
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;

    return 0;
}
